// Copyright 2016, National University of Defense Technology
// Authors: Xuhao Chen <cxh@illinois.edu>
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <fstream>
#include <sstream>
#include <string>
#include <vector>
#include <set>
using namespace std;
#include "common.h"
#include "graph_io.h"
#include "variants.h"

int main(int argc, char *argv[]) {
	printf("Connected Component with CUDA by Xuhao Chen\n");
	if (argc < 2) {
		printf("Usage: %s <graph> [device(0/1)]\n", argv[0]);
		exit(1);
	}
	int m, nnz, *h_row_offsets = NULL, *h_column_indices = NULL;
	foru *h_weight = NULL;
	if (strstr(argv[1], ".mtx"))
		mtx2csr(argv[1], m, nnz, h_row_offsets, h_column_indices, h_weight);
	else if (strstr(argv[1], ".graph"))
		graph2csr(argv[1], m, nnz, h_row_offsets, h_column_indices, h_weight);
	else if (strstr(argv[1], ".gr"))
		gr2csr(argv[1], m, nnz, h_row_offsets, h_column_indices, h_weight);
	else { printf("Unrecognizable input file format\n"); exit(0); }

	int device = 0;
	if (argc > 2) device = atoi(argv[2]);
	assert(device == 0 || device == 1);
	int deviceCount = 0;
	CUDA_SAFE_CALL(hipGetDeviceCount(&deviceCount));
	CUDA_SAFE_CALL(hipSetDevice(device));
	hipDeviceProp_t deviceProp;
	CUDA_SAFE_CALL(hipGetDeviceProperties(&deviceProp, device));
	int nSM = deviceProp.multiProcessorCount;
	fprintf(stdout, "Found %d devices, using device %d (%s), compute capability %d.%d, cores %d*%d.\n", 
			deviceCount, device, deviceProp.name, deviceProp.major, deviceProp.minor, nSM, ConvertSMVer2Cores(deviceProp.major, deviceProp.minor));

	int *h_degree = (int *)malloc(m * sizeof(int));
	for(int i = 0; i < m; i ++) {
		h_degree[i] = h_row_offsets[i + 1] - h_row_offsets[i];
	}
	int *d_row_offsets, *d_column_indices, *d_degree;
	CUDA_SAFE_CALL(hipMalloc((void **)&d_row_offsets, (m + 1) * sizeof(int)));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_column_indices, nnz * sizeof(int)));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_degree, m * sizeof(int)));
	CUDA_SAFE_CALL(hipMemcpy(d_row_offsets, h_row_offsets, (m + 1) * sizeof(int), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_column_indices, h_column_indices, nnz * sizeof(int), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_degree, h_degree, m * sizeof(int), hipMemcpyHostToDevice));

	SGD(m, nnz, d_row_offsets, d_column_indices, d_degree);
	printf("Verifying...\n");
	//SGDVerifier(m, h_row_offsets, h_column_indices);
	CUDA_SAFE_CALL(hipFree(d_row_offsets));
	CUDA_SAFE_CALL(hipFree(d_column_indices));
	CUDA_SAFE_CALL(hipFree(d_degree));
	free(h_row_offsets);
	free(h_column_indices);
	free(h_degree);
	return 0;
}
