#include "hip/hip_runtime.h"
// Copyright 2016, National University of Defense Technology
// Authors: Xuhao Chen <cxh@illinois.edu>
#define PR_VARIANT "pull_lb"
#include "pr.h"
#include "timer.h"
#include "cuda_launch_config.hpp"
#include "cutil_subset.h"
#include <hipcub/hipcub.hpp>
#define FUSED 0
typedef hipcub::BlockReduce<float, BLOCK_SIZE> BlockReduce;
typedef hipcub::BlockScan<int, BLOCK_SIZE> BlockScan;

__global__ void contrib(int m, ScoreT *scores, int *degree, ScoreT *outgoing_contrib) {
	int u = blockIdx.x * blockDim.x + threadIdx.x;
	if (u < m) outgoing_contrib[u] = scores[u] / degree[u];
}

__global__ void l1norm(int m, ScoreT *scores, ScoreT *sums, float *diff, ScoreT base_score) {
	int u = blockIdx.x * blockDim.x + threadIdx.x;
	__shared__ typename BlockReduce::TempStorage temp_storage;
	float local_diff = 0;
	if(u < m) {
		ScoreT new_score = base_score + kDamp * sums[u];
		local_diff += fabs(new_score - scores[u]);
		scores[u] = new_score;
		sums[u] = 0;
	}
	float block_sum = BlockReduce(temp_storage).Sum(local_diff);
	if(threadIdx.x == 0) atomicAdd(diff, block_sum);
}

__device__ __forceinline__ unsigned LaneId() {
	unsigned ret;
	asm("mov.u32 %0, %laneid;" : "=r"(ret));
	return ret;
}

__device__ __forceinline__ void expandByWarp(int m, int *row_offsets, int *column_indices, ScoreT *sums, ScoreT *outgoing_contrib) {
	unsigned id = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned warp_id = threadIdx.x >> LOG_WARP_SIZE;
	unsigned lane_id = LaneId();
	__shared__ int owner[NUM_WARPS];
	__shared__ int sh_vertex[NUM_WARPS];
	owner[warp_id] = -1;
	int size = 0;
	int dst = id;
	if(dst < m) {
		size = row_offsets[dst+1] - row_offsets[dst];
	}
	while(__any(size) >= WARP_SIZE) {
		if(size >= WARP_SIZE)
			owner[warp_id] = lane_id;
		if(owner[warp_id] == lane_id) {
			sh_vertex[warp_id] = dst;
			owner[warp_id] = -1;
			size = 0;
		}
		int winner = sh_vertex[warp_id];
		int row_begin = row_offsets[winner];
		int row_end = row_offsets[winner+1];
		int neighbor_size = row_end - row_begin;
		int num = ((neighbor_size + WARP_SIZE - 1) / WARP_SIZE) * WARP_SIZE;
		for(int i = lane_id; i < num; i+= WARP_SIZE) {
			int edge = row_begin + i;
			if(i < neighbor_size) {
			}
		}
	}
}

__global__ void pull_step(int m, IndexT *row_offsets, IndexT *column_indices, ScoreT *sums, ScoreT *outgoing_contrib) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int tx = threadIdx.x;
	int dst = tid;
	__shared__ BlockScan::TempStorage temp_storage;
	__shared__ int gather_offsets[BLOCK_SIZE];
	__shared__ int dst_id[BLOCK_SIZE];
	__shared__ ScoreT incoming_total[BLOCK_SIZE];
	gather_offsets[tx] = 0;
	dst_id[tx] = 0;
	incoming_total[tx] = 0.0;
	int row_begin = 0, row_end = 0, degree = 0;
	int scratch_offset = 0;
	int total_edges = 0;
	if (dst < m) {
		row_begin = row_offsets[dst];
		row_end = row_offsets[dst+1];
		degree = row_end - row_begin;
	}
	BlockScan(temp_storage).ExclusiveSum(degree, scratch_offset, total_edges);
	int done = 0;
	int neighbors_done = 0;
	int neighbor_offset = 0;
	while (total_edges > 0) {
		__syncthreads();
		int i;
		for(i = 0; neighbors_done + i < degree && (scratch_offset + i - done) < BLOCK_SIZE; i++) {
			int j = scratch_offset + i - done;
			gather_offsets[j] = neighbor_offset + neighbors_done + i;
			dst_id[j] = dst;
		}
		neighbors_done += i;
		scratch_offset += i;
		__syncthreads();
		if(tx < total_edges) {
			int edge = gather_offsets[tx];
			int src = column_indices[edge];
			int dst = dst_id[tx];
			atomicAdd(&incoming_total[dst], outgoing_contrib[src]);
		}
		total_edges -= BLOCK_SIZE;
		done += BLOCK_SIZE;
	}
	sums[dst] = incoming_total[tx];
}

void PRSolver(int m, int nnz, IndexT *in_row_offsets, IndexT *in_column_indices, IndexT *out_row_offsets, IndexT *out_column_indices, int *degrees, ScoreT *scores) {
	//print_device_info(0);
	IndexT *d_row_offsets, *d_column_indices;
	int *d_degrees;
	CUDA_SAFE_CALL(hipMalloc((void **)&d_row_offsets, (m + 1) * sizeof(IndexT)));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_column_indices, nnz * sizeof(IndexT)));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_degrees, m * sizeof(int)));
	CUDA_SAFE_CALL(hipMemcpy(d_row_offsets, in_row_offsets, (m + 1) * sizeof(IndexT), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_column_indices, in_column_indices, nnz * sizeof(IndexT), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_degrees, degrees, m * sizeof(int), hipMemcpyHostToDevice));
	ScoreT *d_scores, *d_sums, *d_contrib;
	CUDA_SAFE_CALL(hipMalloc((void **)&d_scores, m * sizeof(ScoreT)));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_sums, m * sizeof(ScoreT)));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_contrib, m * sizeof(ScoreT)));
	CUDA_SAFE_CALL(hipMemcpy(d_scores, scores, m * sizeof(ScoreT), hipMemcpyHostToDevice));
	float *d_diff, h_diff;
	CUDA_SAFE_CALL(hipMalloc((void **)&d_diff, sizeof(float)));

	int iter = 0;
	int nthreads = BLOCK_SIZE;
	const ScoreT base_score = (1.0f - kDamp) / m;
	int nblocks = (m - 1) / nthreads + 1;
	printf("Launching CUDA PR solver (%d CTAs, %d threads/CTA) ...\n", nblocks, nthreads);

	Timer t;
	t.Start();
	do {
		++iter;
		h_diff = 0;
		CUDA_SAFE_CALL(hipMemcpy(d_diff, &h_diff, sizeof(float), hipMemcpyHostToDevice));
		contrib<<<nblocks, nthreads>>>(m, d_scores, d_degrees, d_contrib);
		CudaTest("solving kernel contrib failed");
		pull_step <<<nblocks, nthreads>>>(m, d_row_offsets, d_column_indices, d_sums, d_contrib);
		l1norm <<<nblocks, nthreads>>> (m, d_scores, d_sums, d_diff, base_score);
		CudaTest("solving kernel pull failed");
		CUDA_SAFE_CALL(hipMemcpy(&h_diff, d_diff, sizeof(float), hipMemcpyDeviceToHost));
		printf(" %2d    %f\n", iter, h_diff);
	} while (h_diff > EPSILON && iter < MAX_ITER);
	CUDA_SAFE_CALL(hipDeviceSynchronize());
	t.Stop();

	printf("\titerations = %d.\n", iter);
	printf("\truntime [%s] = %f ms.\n", PR_VARIANT, t.Millisecs());
	CUDA_SAFE_CALL(hipMemcpy(scores, d_scores, m * sizeof(ScoreT), hipMemcpyDeviceToHost));
	CUDA_SAFE_CALL(hipFree(d_row_offsets));
	CUDA_SAFE_CALL(hipFree(d_column_indices));
	CUDA_SAFE_CALL(hipFree(d_degrees));
	CUDA_SAFE_CALL(hipFree(d_scores));
	CUDA_SAFE_CALL(hipFree(d_sums));
	CUDA_SAFE_CALL(hipFree(d_contrib));
	CUDA_SAFE_CALL(hipFree(d_diff));
	return;
}
