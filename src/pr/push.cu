#include "hip/hip_runtime.h"
// Copyright 2016, National University of Defense Technology
// Authors: Xuhao Chen <cxh@illinois.edu>
#define PR_VARIANT "push"
#include "pr.h"
#include "timer.h"
#include "cuda_launch_config.hpp"
#include "cutil_subset.h"
#include <hipcub/hipcub.hpp>
typedef hipcub::BlockReduce<float, BLOCK_SIZE> BlockReduce;
typedef hipcub::BlockScan<int, BLOCK_SIZE> BlockScan;

__global__ void initialize(int m, ScoreT *sums, ScoreT base_score) {
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < m) sums[id] = 0;
}

#if 0
__global__ void push_step(int m, int *row_offsets, int *column_indices, ScoreT *scores, ScoreT *sums, bool *processed) {
	int src = blockIdx.x * blockDim.x + threadIdx.x;
	if(src < m) {
		int row_begin = row_offsets[src];
		int row_end = row_offsets[src + 1];
		int degree = row_end - row_begin;
		ScoreT value = scores[src] / (ScoreT)degree;
		for (int offset = row_begin; offset < row_end; ++ offset) {
			int dst = column_indices[offset];
			atomicAdd(&sums[dst], value);
		}
	}
}
#else
__global__ void push_step(int m, IndexT *row_offsets, IndexT *column_indices, ScoreT *scores, ScoreT *sums, bool *processed) {
	//expandByCta(m, row_offsets, column_indices, scores, sums, processed);
	//expandByWarp(m, row_offsets, column_indices, scores, sums, processed);
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int tx = threadIdx.x;
	int src = tid;
	__shared__ BlockScan::TempStorage temp_storage;
	__shared__ int gather_offsets[BLOCK_SIZE];
	__shared__ int src_idx[BLOCK_SIZE];
	__shared__ ScoreT value[BLOCK_SIZE];
	gather_offsets[tx] = 0;
	src_idx[tx] = 0;
	value[tx] = 0;
	int row_begin = 0, row_end = 0, degree = 0;
	int scratch_offset = 0;
	int total_edges = 0;
	if (src < m && !processed[src]) {
		row_begin = row_offsets[src];
		row_end = row_offsets[src+1];
		degree = row_end - row_begin;
		if (degree > 0) value[tx] = scores[src] / (ScoreT)degree;
	}
	BlockScan(temp_storage).ExclusiveSum(degree, scratch_offset, total_edges);
	int done = 0;
	int neighbors_done = 0;
	int neighbor_offset = 0;
	while (total_edges > 0) {
		__syncthreads();
		int i;
		for(i = 0; neighbors_done + i < degree && (scratch_offset + i - done) < BLOCK_SIZE; i++) {
			int j = scratch_offset + i - done;
			gather_offsets[j] = neighbor_offset + neighbors_done + i;
			src_idx[j] = tx;
		}
		neighbors_done += i;
		scratch_offset += i;
		__syncthreads();
		if(tx < total_edges) {
			int edge = gather_offsets[tx];
			int dst = column_indices[edge];
			atomicAdd(&sums[dst], value[src_idx[tx]]);
		}
		total_edges -= BLOCK_SIZE;
		done += BLOCK_SIZE;
	}
}
#endif

__global__ void l1norm(int m, ScoreT *scores, ScoreT *sums, float *diff, ScoreT base_score) {
	int u = blockIdx.x * blockDim.x + threadIdx.x;
	__shared__ typename BlockReduce::TempStorage temp_storage;
	float local_diff = 0;
	if(u < m) {
		ScoreT new_score = base_score + kDamp * sums[u];
		local_diff += fabs(new_score - scores[u]);
		scores[u] = new_score;
		sums[u] = 0;
	}
	float block_sum = BlockReduce(temp_storage).Sum(local_diff);
	if(threadIdx.x == 0) atomicAdd(diff, block_sum);
}

void PRSolver(int m, int nnz, int *in_row_offsets, int *in_column_indices, int *h_row_offsets, int *h_column_indices, int *h_degree, ScoreT *h_scores) {
	int *d_row_offsets, *d_column_indices;
	ScoreT *d_scores;
	CUDA_SAFE_CALL(hipMalloc((void **)&d_row_offsets, (m + 1) * sizeof(int)));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_column_indices, nnz * sizeof(int)));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_scores, m * sizeof(ScoreT)));
	CUDA_SAFE_CALL(hipMemcpy(d_row_offsets, h_row_offsets, (m + 1) * sizeof(int), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_column_indices, h_column_indices, nnz * sizeof(int), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_scores, h_scores, m * sizeof(ScoreT), hipMemcpyHostToDevice));
	ScoreT *d_sums;
	CUDA_SAFE_CALL(hipMalloc((void **)&d_sums, m * sizeof(ScoreT)));
	float *d_diff, h_diff;
	CUDA_SAFE_CALL(hipMalloc((void **)&d_diff, sizeof(float)));

	bool *d_processed;
	CUDA_SAFE_CALL(hipMalloc((void **)&d_processed, m * sizeof(bool)));

	Timer t;
	int iter = 0;
	const ScoreT base_score = (1.0f - kDamp) / m;
	int nthreads = BLOCK_SIZE;
	int nblocks = (m - 1) / nthreads + 1;
	initialize <<<nblocks, nthreads>>> (m, d_sums, base_score);
	CudaTest("initializing failed");
	printf("Launching CUDA PR solver (%d CTAs, %d threads/CTA) ...\n", nblocks, nthreads);

	t.Start();
	do {
		++ iter;
		h_diff = 0;
		CUDA_SAFE_CALL(hipMemcpy(d_diff, &h_diff, sizeof(float), hipMemcpyHostToDevice));
		CUDA_SAFE_CALL(hipMemset(d_processed, 0, m * sizeof(bool)));
		push_step <<<nblocks, nthreads>>> (m, d_row_offsets, d_column_indices, d_scores, d_sums, d_processed);
		CudaTest("solving kernel push failed");
		l1norm <<<nblocks, nthreads>>> (m, d_scores, d_sums, d_diff, base_score);
		CudaTest("solving kernel reduce failed");
		CUDA_SAFE_CALL(hipMemcpy(&h_diff, d_diff, sizeof(float), hipMemcpyDeviceToHost));
		//printf("iteration=%d, diff=%f\n", iter, h_diff);
		printf(" %2d    %lf\n", iter, h_diff);
		//CUDA_SAFE_CALL(hipMemcpy(h_scores, d_scores, m * sizeof(ScoreT), hipMemcpyDeviceToHost));
	} while (h_diff > EPSILON && iter < MAX_ITER);
	CUDA_SAFE_CALL(hipDeviceSynchronize());
	t.Stop();
	printf("\titerations = %d.\n", iter);
	printf("\truntime [%s] = %f ms.\n", PR_VARIANT, t.Millisecs());
	CUDA_SAFE_CALL(hipMemcpy(h_scores, d_scores, m * sizeof(ScoreT), hipMemcpyDeviceToHost));
	CUDA_SAFE_CALL(hipFree(d_row_offsets));
	CUDA_SAFE_CALL(hipFree(d_column_indices));
	CUDA_SAFE_CALL(hipFree(d_scores));
	CUDA_SAFE_CALL(hipFree(d_sums));
	CUDA_SAFE_CALL(hipFree(d_diff));
	return;
}
