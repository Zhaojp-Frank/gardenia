#include "hip/hip_runtime.h"
#define PR_VARIANT "fusion"
#include <hipcub/hipcub.hpp>
#include "pr.h"
#include "timer.h"
#include "gbar.h"
#include "cuda_launch_config.hpp"
#include "cutil_subset.h"
typedef hipcub::BlockReduce<float, BLKSIZE> BlockReduce;

__global__ void initialize(int m, ScoreT *scores, ScoreT init_score) {
	unsigned id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < m) {
		scores[id] = init_score;
		//active[id] = true;
	}
}

__global__ void pr_kernel(int m, int *row_offsets, int *column_indices, ScoreT *scores, int *degree, ScoreT *outgoing_contrib, float *diff, bool *active, ScoreT base_score, GlobalBarrier gb) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int num_vertices_per_thread = (m - 1) / (gridDim.x * blockDim.x) + 1;
	int total_inputs = num_vertices_per_thread;
	for (int src = tid; total_inputs > 0; src += blockDim.x * gridDim.x, total_inputs--) {
		if (src < m) {
			outgoing_contrib[src] = scores[src] / degree[src];
		}
	}
	gb.Sync();
	float local_diff = 0.0f;
	total_inputs = num_vertices_per_thread;
	__shared__ typename BlockReduce::TempStorage temp_storage;
	for (int src = tid; total_inputs > 0; src += blockDim.x * gridDim.x, total_inputs--) {
		if (src < m) {
			int row_begin = row_offsets[src];
			int row_end = row_offsets[src + 1];
			ScoreT incoming_total = 0;
			for (int offset = row_begin; offset < row_end; ++ offset) {
				int dst = column_indices[offset];
				incoming_total += outgoing_contrib[dst];
			}
			ScoreT old_score = scores[src];
			scores[src] = base_score + kDamp * incoming_total;
			local_diff += abs(scores[src] - old_score);
		}
	}
	float block_sum = BlockReduce(temp_storage).Sum(local_diff);
	if(threadIdx.x == 0) {
		atomicAdd(diff, block_sum);
	}
}

void PRSolver(int m, int nnz, int *h_row_offsets, int *h_column_indices, int *out_row_offsets, int *out_column_indices, int *h_degree, ScoreT *h_scores) {
	float *d_diff, h_diff;
	ScoreT *d_contrib;
	bool *d_active;
	Timer t;
	int iter = 0;
	int nthreads = BLKSIZE;
	int nblocks = (m - 1) / nthreads + 1;

	int *d_row_offsets, *d_column_indices, *d_degree;
	ScoreT *d_scores;
	CUDA_SAFE_CALL(hipMalloc((void **)&d_row_offsets, (m + 1) * sizeof(int)));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_column_indices, nnz * sizeof(int)));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_degree, m * sizeof(int)));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_scores, m * sizeof(ScoreT)));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_contrib, m * sizeof(ScoreT)));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_diff, sizeof(float)));
	//CUDA_SAFE_CALL(hipMalloc((void **)&d_active, m * sizeof(bool)));
	CUDA_SAFE_CALL(hipMemcpy(d_row_offsets, h_row_offsets, (m + 1) * sizeof(int), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_column_indices, h_column_indices, nnz * sizeof(int), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_degree, h_degree, m * sizeof(int), hipMemcpyHostToDevice));

	const ScoreT base_score = (1.0f - kDamp) / m;
	const ScoreT init_score = 1.0f / m;
	initialize <<<nblocks, nthreads>>> (m, d_scores, init_score);
	CudaTest("initializing failed");

	int max_blocks = 5;
	max_blocks = maximum_residency(pr_kernel, nthreads, 0);
    hipDeviceProp_t deviceProp;
	CUDA_SAFE_CALL(hipGetDeviceProperties(&deviceProp, 0));
	int nSM = deviceProp.multiProcessorCount;
	nblocks = nSM * max_blocks;
	GlobalBarrierLifetime gb;
	gb.Setup(nblocks);
	printf("Solving, max_blocks=%d, nblocks=%d, nthreads=%d\n", max_blocks, nblocks, nthreads);
	t.Start();
	do {
		++iter;
		h_diff = 0.0f;
		CUDA_SAFE_CALL(hipMemcpy(d_diff, &h_diff, sizeof(h_diff), hipMemcpyHostToDevice));
		pr_kernel<<<nblocks, nthreads>>>(m, d_row_offsets, d_column_indices, d_scores, d_degree, d_contrib, d_diff, d_active, base_score, gb);
		CudaTest("solving kernel failed");
		CUDA_SAFE_CALL(hipMemcpy(&h_diff, d_diff, sizeof(h_diff), hipMemcpyDeviceToHost));
		printf("iteration=%d, diff=%f\n", iter, h_diff);
	} while (h_diff > EPSILON && iter < MAX_ITER);
	CUDA_SAFE_CALL(hipDeviceSynchronize());
	t.Stop();
	printf("\titerations = %d.\n", iter);
	printf("\truntime [%s] = %f ms.\n", PR_VARIANT, t.Millisecs());
	CUDA_SAFE_CALL(hipMemcpy(h_scores, d_scores, m * sizeof(ScoreT), hipMemcpyDeviceToHost));
	CUDA_SAFE_CALL(hipFree(d_row_offsets));
	CUDA_SAFE_CALL(hipFree(d_column_indices));
	CUDA_SAFE_CALL(hipFree(d_degree));
	CUDA_SAFE_CALL(hipFree(d_scores));
	CUDA_SAFE_CALL(hipFree(d_contrib));
	CUDA_SAFE_CALL(hipFree(d_diff));
	//CUDA_SAFE_CALL(hipFree(d_active));
	return;
}
