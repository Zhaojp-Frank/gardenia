#include "hip/hip_runtime.h"
#define PR_VARIANT "gather"
#include <hipcub/hipcub.hpp>
#include "pr.h"
#include "timer.h"
#include "cuda_launch_config.hpp"
#include "cutil_subset.h"
typedef hipcub::BlockReduce<float, BLKSIZE> BlockReduce;

__global__ void initialize(int m, ScoreT *score, ScoreT init_score) {
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < m) {
		score[id] = init_score;
		//active[id] = true;
	}
}

__global__ void process(int m, ScoreT *score, int *degree, ScoreT *outgoing_contrib, bool *active) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int total_inputs = (m - 1) / (gridDim.x * blockDim.x) + 1;
	for (int src = tid; total_inputs > 0; src += blockDim.x * gridDim.x, total_inputs--) {
		if (src < m) {
		//if (src < m && active[src]) {
			outgoing_contrib[src] = score[src] / degree[src];
			//if(src<3) printf("score[%d]=%.8f, degree[%d]=%d\n", src, score[src], src, degree[src]);
		}
	}
}

// gather operation needs incoming neighbor list
__global__ void gather(int m, int *row_offsets, int *column_indices, ScoreT *score, ScoreT *contrib, float *diff, bool *active, const ScoreT base_score) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	__shared__ typename BlockReduce::TempStorage temp_storage;
	int total_inputs = (m - 1) / (gridDim.x * blockDim.x) + 1;
	float error = 0;
	for (int src = tid; total_inputs > 0; src += blockDim.x * gridDim.x, total_inputs--) {
		if (src < m) {
		//if (src < m && active[src]) {
			int row_begin = row_offsets[src];
			int row_end = row_offsets[src + 1];
			ScoreT incoming_total = 0;
			for (int offset = row_begin; offset < row_end; ++ offset) {
				int dst = column_indices[offset];
				//if(src==0) printf("dst=%d\n", dst);
				incoming_total += contrib[dst];
			}
			ScoreT old_score = score[src];
			score[src] = base_score + kDamp * incoming_total;
			error += fabs(score[src] - old_score);
		}
	}
	float block_sum = BlockReduce(temp_storage).Sum(error);
	if(threadIdx.x == 0) atomicAdd(diff, block_sum);
}

void PRSolver(int m, int nnz, int *h_row_offsets, int *h_column_indices, int *out_row_offsets, int *out_column_indices, int *h_degree, ScoreT *h_score) {
	print_device_info(0);
	Timer t;
	float *d_diff, h_diff;
	ScoreT *d_contrib;
	bool *d_active;
	int iter = 0;
	int nthreads = BLKSIZE;
	int nblocks = (m - 1) / nthreads + 1;

	int *d_row_offsets, *d_column_indices, *d_degree;
	ScoreT *d_score;
	CUDA_SAFE_CALL(hipMalloc((void **)&d_row_offsets, (m + 1) * sizeof(int)));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_column_indices, nnz * sizeof(int)));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_degree, m * sizeof(int)));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_score, m * sizeof(ScoreT)));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_contrib, m * sizeof(ScoreT)));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_diff, sizeof(float)));
	//CUDA_SAFE_CALL(hipMalloc((void **)&d_active, m * sizeof(bool)));
	CUDA_SAFE_CALL(hipMemcpy(d_row_offsets, h_row_offsets, (m + 1) * sizeof(int), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_column_indices, h_column_indices, nnz * sizeof(int), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_degree, h_degree, m * sizeof(int), hipMemcpyHostToDevice));
	const ScoreT base_score = (1.0f - kDamp) / m;
	const ScoreT init_score = 1.0f / m;
	printf("base_score=%.8f, init_score=%.8f\n", base_score, init_score);
	initialize <<<nblocks, nthreads>>> (m, d_score, init_score);
	CudaTest("initializing failed");
	//CUDA_SAFE_CALL(hipMemcpy(h_score, d_score, m * sizeof(ScoreT), hipMemcpyDeviceToHost));
	//for(int i = 0; i < 5; i++) printf("score[%d]=%.8f\n", i, h_score[i]);

	size_t max_blocks = 5;
	max_blocks = maximum_residency(gather, nthreads, 0);
	printf("Solving, max_blocks=%d, nblocks=%d, nthreads=%d\n", max_blocks, nblocks, nthreads);
	t.Start();
	do {
		++iter;
		h_diff = 0;
		CUDA_SAFE_CALL(hipMemcpy(d_diff, &h_diff, sizeof(float), hipMemcpyHostToDevice));
		process<<<nblocks, nthreads>>>(m, d_score, d_degree, d_contrib, d_active);
		CudaTest("solving kernel1 failed");
		gather<<<nblocks, nthreads>>>(m, d_row_offsets, d_column_indices, d_score, d_contrib, d_diff, d_active, base_score);
		CudaTest("solving kernel2 failed");
		CUDA_SAFE_CALL(hipMemcpy(&h_diff, d_diff, sizeof(float), hipMemcpyDeviceToHost));
		printf("iteration=%d, diff=%f\n", iter, h_diff);
		//CUDA_SAFE_CALL(hipMemcpy(h_score, d_score, m * sizeof(ScoreT), hipMemcpyDeviceToHost));
		//for(int i = 0; i < 5; i++) printf("score[%d]=%.8f\n", i, h_score[i]);
	} while (h_diff > EPSILON && iter < MAX_ITER);
	CUDA_SAFE_CALL(hipDeviceSynchronize());
	t.Stop();
	printf("\titerations = %d.\n", iter);
	printf("\truntime [%s] = %f ms.\n", PR_VARIANT, t.Millisecs());
	CUDA_SAFE_CALL(hipMemcpy(h_score, d_score, m * sizeof(ScoreT), hipMemcpyDeviceToHost));
	CUDA_SAFE_CALL(hipFree(d_row_offsets));
	CUDA_SAFE_CALL(hipFree(d_column_indices));
	CUDA_SAFE_CALL(hipFree(d_degree));
	CUDA_SAFE_CALL(hipFree(d_score));
	CUDA_SAFE_CALL(hipFree(d_contrib));
	CUDA_SAFE_CALL(hipFree(d_diff));
	//CUDA_SAFE_CALL(hipFree(d_active));
	return;
}
