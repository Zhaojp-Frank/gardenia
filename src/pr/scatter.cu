#include "hip/hip_runtime.h"
// Copyright 2016, National University of Defense Technology
// Authors: Xuhao Chen <cxh@illinois.edu>
#define PR_VARIANT "scatter"
#include "pr.h"
#include "timer.h"
#include "cuda_launch_config.hpp"
#include "cutil_subset.h"
#include <hipcub/hipcub.hpp>
typedef hipcub::BlockReduce<float, BLKSIZE> BlockReduce;

__global__ void initialize(int m, ScoreT *cur_scores, ScoreT *next_scores, ScoreT init_score, ScoreT base_score) {
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < m) {
		cur_scores[id] = init_score;
		next_scores[id] = base_score;
	}
}

__global__ void scatter(int m, int *row_offsets, int *column_indices, ScoreT *cur_scores, ScoreT *next_scores) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int total_inputs = (m - 1) / (gridDim.x * blockDim.x) + 1;
	for (int src = tid; total_inputs > 0; src += blockDim.x * gridDim.x, total_inputs--) {
		if(src < m) {
			int row_begin = row_offsets[src];
			int row_end = row_offsets[src + 1];
			int degree = row_end - row_begin;
			ScoreT value = kDamp * cur_scores[src] / (ScoreT)degree;
			for (int offset = row_begin; offset < row_end; ++ offset) {
				int dst = column_indices[offset];
				atomicAdd(&next_scores[dst], value);
			}
		}
	}
}

__global__ void reduce(int m, ScoreT *cur_scores, ScoreT *next_scores, float *diff) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	__shared__ typename BlockReduce::TempStorage temp_storage;
	int total_inputs = (m - 1) / (gridDim.x * blockDim.x) + 1;
	float local_diff = 0;
	for (int src = tid; total_inputs > 0; src += blockDim.x * gridDim.x, total_inputs--) {
		if(src < m) {
			local_diff += fabs(next_scores[src] - cur_scores[src]);
			cur_scores[src] = next_scores[src];
			next_scores[src] = (1.0f - kDamp) / m;
		}
	}
	float block_sum = BlockReduce(temp_storage).Sum(local_diff);
	if(threadIdx.x == 0) atomicAdd(diff, block_sum);
}

void PRSolver(int m, int nnz, int *in_row_offsets, int *in_column_indices, int *h_row_offsets, int *h_column_indices, int *h_degree, ScoreT *h_scores) {
	float *d_diff, h_diff;
	Timer t;
	ScoreT *d_next_scores;
	int iter = 0;
	int nthreads = BLKSIZE;
	int nblocks = (m - 1) / nthreads + 1;

	int *d_row_offsets, *d_column_indices;
	ScoreT *d_scores;
	CUDA_SAFE_CALL(hipMalloc((void **)&d_row_offsets, (m + 1) * sizeof(int)));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_column_indices, nnz * sizeof(int)));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_scores, m * sizeof(ScoreT)));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_next_scores, m * sizeof(ScoreT)));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_diff, sizeof(float)));
	CUDA_SAFE_CALL(hipMemcpy(d_row_offsets, h_row_offsets, (m + 1) * sizeof(int), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_column_indices, h_column_indices, nnz * sizeof(int), hipMemcpyHostToDevice));
	const ScoreT init_score = 1.0f / m;
	const ScoreT base_score = (1.0f - kDamp) / m;
	printf("base_score=%.8f, init_score=%.8f\n", base_score, init_score);
	initialize <<<nblocks, nthreads>>> (m, d_scores, d_next_scores, init_score, base_score);
	CudaTest("initializing failed");
	//CUDA_SAFE_CALL(hipMemcpy(h_scores, d_scores, m * sizeof(ScoreT), hipMemcpyDeviceToHost));

	int max_blocks = maximum_residency(scatter, nthreads, 0);
	//const size_t max_blocks = 5;
	printf("Solving, max_blocks=%d, nblocks=%d, nthreads=%d\n", max_blocks, nblocks, nthreads);
	t.Start();
	do {
		++ iter;
		h_diff = 0;
		CUDA_SAFE_CALL(hipMemcpy(d_diff, &h_diff, sizeof(float), hipMemcpyHostToDevice));
		scatter <<<nblocks, nthreads>>> (m, d_row_offsets, d_column_indices, d_scores, d_next_scores);
		CudaTest("solving kernel1 failed");
		reduce <<<nblocks, nthreads>>> (m, d_scores, d_next_scores, d_diff);
		CudaTest("solving kernel2 failed");
		CUDA_SAFE_CALL(hipMemcpy(&h_diff, d_diff, sizeof(float), hipMemcpyDeviceToHost));
		printf("iteration=%d, diff=%f\n", iter, h_diff);
		//CUDA_SAFE_CALL(hipMemcpy(h_scores, d_scores, m * sizeof(ScoreT), hipMemcpyDeviceToHost));
	} while (h_diff > EPSILON && iter < MAX_ITER);
	CUDA_SAFE_CALL(hipDeviceSynchronize());
	t.Stop();
	printf("\titerations = %d.\n", iter);
	printf("\truntime [%s] = %f ms.\n", PR_VARIANT, t.Millisecs());
	CUDA_SAFE_CALL(hipMemcpy(h_scores, d_scores, m * sizeof(ScoreT), hipMemcpyDeviceToHost));
	CUDA_SAFE_CALL(hipFree(d_row_offsets));
	CUDA_SAFE_CALL(hipFree(d_column_indices));
	CUDA_SAFE_CALL(hipFree(d_scores));
	CUDA_SAFE_CALL(hipFree(d_next_scores));
	CUDA_SAFE_CALL(hipFree(d_diff));
	return;
}
