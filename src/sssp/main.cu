// Copyright 2016, National University of Defense Technology
// Authors: Xuhao Chen <cxh@illinois.edu>
#include "common.h"
#include "graph_io.h"
#include "variants.h"
#include "verifier.h"

int main(int argc, char *argv[]) {
	printf("Single Source Shortest Path (SSSP) with CUDA by Xuhao Chen\n");
	if (argc < 2) {
		printf("Usage: %s <graph> [device(0/1)]\n", argv[0]);
		exit(1);
	}

	// CSR data structures
	int m, nnz, *h_row_offsets = NULL, *h_column_indices = NULL, *h_degree = NULL;
	W_TYPE *h_weight = NULL;
	read_graph(argc, argv, m, nnz, h_row_offsets, h_column_indices, h_degree, h_weight, false);
	int *d_row_offsets, *d_column_indices;
	W_TYPE *d_weight;
	CUDA_SAFE_CALL(hipMalloc((void **)&d_row_offsets, (m + 1) * sizeof(int)));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_column_indices, nnz * sizeof(int)));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_weight, nnz * sizeof(W_TYPE)));
	CUDA_SAFE_CALL(hipMemcpy(d_row_offsets, h_row_offsets, (m + 1) * sizeof(int), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_column_indices, h_column_indices, nnz * sizeof(int), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_weight, h_weight, nnz * sizeof(W_TYPE), hipMemcpyHostToDevice));

	// distance array
	DistT *h_dist = (DistT *) malloc(m * sizeof(DistT));
	for(int i = 0; i < m; i ++) {
		h_dist[i] = MYINFINITY;
	}
	DistT * d_dist;
	CUDA_SAFE_CALL(hipMalloc((void **)&d_dist, m * sizeof(DistT)));
	CUDA_SAFE_CALL(hipMemcpy(d_dist, h_dist, m * sizeof(DistT), hipMemcpyHostToDevice));

	SSSPSolver(m, nnz, d_row_offsets, d_column_indices, d_weight, d_dist);
	CUDA_SAFE_CALL(hipMemcpy(h_dist, d_dist, m * sizeof(DistT), hipMemcpyDeviceToHost));
	SSSPVerifier(m, h_row_offsets, h_column_indices, h_weight, h_dist);
	//write_solution("sssp-out.txt", m, h_dist);
	CUDA_SAFE_CALL(hipFree(d_row_offsets));
	CUDA_SAFE_CALL(hipFree(d_column_indices));
	CUDA_SAFE_CALL(hipFree(d_weight));
	CUDA_SAFE_CALL(hipFree(d_dist));
	free(h_row_offsets);
	free(h_column_indices);
	free(h_weight);
	free(h_dist);
	if(h_degree) free(h_degree);
	return 0;
}
