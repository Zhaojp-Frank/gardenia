#include "hip/hip_runtime.h"
// Copyright 2016, National University of Defense Technology
// Authors: Xuhao Chen <cxh@illinois.edu> and Pingfan Li <lipingfan@163.com>
#define COLOR_VARIANT "base"
#include <hipcub/hipcub.hpp>
#include "vc.h"
#include "timer.h"
#include "cuda_launch_config.hpp"
#include "cutil_subset.h"
//#include <thrust/sequence.h>
//#include <thrust/reduce.h>
//#include <thrust/execution_policy.h>
#include "worklistc.h"
typedef hipcub::BlockScan<int, BLKSIZE> BlockScan;

__global__ void initialize(int m, int *colors) {
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < m) {
		colors[id] = MAXCOLOR;
	}   
}

__global__ void first_fit(int m, int *row_offsets, int *column_indices, Worklist2 inwl, int *colors) {
	int id = blockIdx.x * blockDim.x + threadIdx.x;	
	bool forbiddenColors[MAXCOLOR+1];
	int vertex;
	if (inwl.pop_id(id, vertex)) {
		int row_begin = row_offsets[vertex];
		int row_end = row_offsets[vertex + 1];
		for (int j = 0; j < MAXCOLOR; j++)
			forbiddenColors[j] = false;
		for (int offset = row_begin; offset < row_end; offset ++) {
			int neighbor = column_indices[offset];
			int color = colors[neighbor];
			if(color != MAXCOLOR)
				forbiddenColors[color] = true;
		}
		int vertex_color;
		for (vertex_color = 0; vertex_color < MAXCOLOR; vertex_color++) {
			if (!forbiddenColors[vertex_color]) {
				colors[vertex] = vertex_color;
				break;
			}
		}
		assert(vertex_color < MAXCOLOR);
	}
}

__global__ void conflict_resolve(int m, int *row_offsets, int *column_indices, Worklist2 inwl, Worklist2 outwl, int *colors) {
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	int conflicted = 0;
	int vertex;
	if (inwl.pop_id(id, vertex)) {
		int row_begin = row_offsets[vertex];
		int row_end = row_offsets[vertex + 1];
		for (int offset = row_begin; offset < row_end; offset ++) {
			int neighbor = column_indices[offset];
			if (colors[vertex] == colors[neighbor] && vertex < neighbor) {
				conflicted = 1;
				colors[vertex] = MAXCOLOR;
				break;
			}
		}
	}
	//outwl.push_1item<BlockScan>(conflicted, vertex, BLKSIZE);
	if(conflicted) outwl.push(vertex);
}

void VCSolver(int m, int nnz, int *row_offsets, int *column_indices, int *colors) {
	int num_colors = 0, iter = 0;
	Timer t;
	int *d_row_offsets, *d_column_indices, *d_colors;
	for(int i = 0; i < m; i ++) {
		colors[i] = MAXCOLOR;
	}
	
	CUDA_SAFE_CALL(hipMalloc((void **)&d_row_offsets, (m + 1) * sizeof(int)));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_column_indices, nnz * sizeof(int)));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_colors, m * sizeof(int)));
	CUDA_SAFE_CALL(hipMemcpy(d_row_offsets, row_offsets, (m + 1) * sizeof(int), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_column_indices, column_indices, nnz * sizeof(int), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_colors, colors, m * sizeof(int), hipMemcpyHostToDevice));

	Worklist2 inwl(m), outwl(m);
	Worklist2 *inwlptr = &inwl, *outwlptr = &outwl;
	for(int i = 0; i < m; i ++) {
		inwl.wl[i] = i;
	}

	t.Start();
	int nitems = m;
	CUDA_SAFE_CALL(hipMemcpy(inwl.dindex, &m, sizeof(int), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(inwl.dwl, inwl.wl, m * sizeof(int), hipMemcpyHostToDevice));
	//thrust::sequence(thrust::device, inwl.dwl, inwl.dwl + m);
	CUDA_SAFE_CALL(hipDeviceSynchronize());
	while (nitems > 0) {
		iter ++;
		int nblocks = (nitems - 1) / BLKSIZE + 1;
		first_fit<<<nblocks, BLKSIZE>>>(m, d_row_offsets, d_column_indices, *inwlptr, d_colors);
		conflict_resolve<<<nblocks, BLKSIZE>>>(m, d_row_offsets, d_column_indices, *inwlptr, *outwlptr, d_colors);
		nitems = outwlptr->nitems();
		Worklist2 * tmp = inwlptr;
		inwlptr = outwlptr;
		outwlptr = tmp;
		outwlptr->reset();
	}
	CUDA_SAFE_CALL(hipDeviceSynchronize());
	t.Stop();
	CUDA_SAFE_CALL(hipMemcpy(colors, d_colors, m * sizeof(int), hipMemcpyDeviceToHost));
	//num_colors = thrust::reduce(colors, colors + m, 0, thrust::maximum<int>()) + 1;
	//num_colors = thrust::reduce(thrust::device, d_colors, d_colors + m, 0, thrust::maximum<int>()) + 1;
	#pragma omp parallel for reduction(max : num_colors)
	for (int n = 0; n < m; n ++)
		num_colors = max(num_colors, colors[n]);
	
    printf("\titerations = %d.\n", iter);
    printf("\truntime[%s] = %f ms, num_colors = %d.\n", COLOR_VARIANT, t.Millisecs(), num_colors);
	CUDA_SAFE_CALL(hipFree(d_row_offsets));
	CUDA_SAFE_CALL(hipFree(d_column_indices));
	CUDA_SAFE_CALL(hipFree(d_colors));
}

