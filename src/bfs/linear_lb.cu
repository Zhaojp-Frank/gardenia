#include "hip/hip_runtime.h"
// Copyright 2016, National University of Defense Technology
// Authors: Xuhao Chen <cxh@illinois.edu>
#define BFS_VARIANT "linear"
#include "bfs.h"
#include "worklistc.h"
#include "cuda_launch_config.hpp"
#include "cutil_subset.h"
#include <hipcub/hipcub.hpp>
#include "timer.h"

__global__ void initialize(DistT *dist, unsigned int m) {
	unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < m) {
		dist[id] = MYINFINITY;
	}
}

typedef hipcub::BlockScan<int, BLKSIZE> BlockScan;
__device__ void expandByCta(int m, int *row_offsets, int *column_indices, DistT *dist, Worklist2 &in_queue, Worklist2 &out_queue, int depth) {
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	int vertex;
	__shared__ int owner;
	__shared__ int sh_vertex;
	owner = -1;
	int size = 0;
	if(in_queue.pop_id(id, vertex)) {
		size = row_offsets[vertex + 1] - row_offsets[vertex];
	}
	while(true) {
		if(size > BLKSIZE)
			owner = threadIdx.x;
		__syncthreads();
		if(owner == -1)
			break;
		__syncthreads();
		if(owner == threadIdx.x) {
			sh_vertex = vertex;
			in_queue.d_queue[id] = -1;
			owner = -1;
			size = 0;
		}
		__syncthreads();
		int row_begin = row_offsets[sh_vertex];
		int row_end = row_offsets[sh_vertex + 1];
		int neighbor_size = row_end - row_begin;
		int num = ((neighbor_size + blockDim.x - 1) / blockDim.x) * blockDim.x;
		for(int i = threadIdx.x; i < num; i += blockDim.x) {
			//int ncnt = 0;
			int dst = 0;
			int edge = row_begin + i;
			if(i < neighbor_size) {
				dst = column_indices[edge];
				assert(dst < m);
				if(dist[dst] == MYINFINITY) {
					dist[dst] = depth;
					out_queue.push(dst);
					//ncnt = 1;
				}
			}
			//out_queue.push_1item<BlockScan>(ncnt, dst, BLKSIZE);
		}
	}
}

__device__ __forceinline__ unsigned LaneId() {
	unsigned ret;
	asm("mov.u32 %0, %laneid;" : "=r"(ret));
	return ret;
}

#define WARP_SIZE 32
#define LOG_WARP_SIZE 5
#define NUM_WARPS (BLKSIZE / WARP_SIZE)
__device__ __forceinline__ void expandByWarp(int m, int *row_offsets, int *column_indices, DistT *dist, Worklist2 &in_queue, Worklist2 &out_queue, int depth) {
	unsigned id = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned warp_id = threadIdx.x >> LOG_WARP_SIZE;
	unsigned lane_id = LaneId();
	__shared__ int owner[NUM_WARPS];
	__shared__ int sh_vertex[NUM_WARPS];
	owner[warp_id] = -1;
	int size = 0;
	int vertex;
	if(in_queue.pop_id(id, vertex)) {
		if (vertex != -1)
			size = row_offsets[vertex + 1] - row_offsets[vertex];
	}
	while(__any(size) >= WARP_SIZE) {
		if(size >= WARP_SIZE)
			owner[warp_id] = lane_id;
		if(owner[warp_id] == lane_id) {
			sh_vertex[warp_id] = vertex;
			in_queue.d_queue[id] = -1;
			owner[warp_id] = -1;
			size = 0;
		}
		int winner = sh_vertex[warp_id];
		int row_begin = row_offsets[winner];
		int row_end = row_offsets[winner + 1];
		int neighbor_size = row_end - row_begin;
		int num = ((neighbor_size + WARP_SIZE - 1) / WARP_SIZE) * WARP_SIZE;
		for(int i = lane_id; i < num; i+= WARP_SIZE) {
			//int ncnt = 0;
			int dst = 0;
			int edge = row_begin + i;
			if(i < neighbor_size) {
				dst = column_indices[edge];
				assert(dst < m);
				if(dist[dst] == MYINFINITY) {
					dist[dst] = depth;
					out_queue.push(dst);
					//ncnt = 1;
				}
			}
			//out_queue.push_1item<BlockScan>(ncnt, dst, BLKSIZE);
		}
	}
}

__global__ void bfs_kernel(int m, int *row_offsets, int *column_indices, DistT *dist, Worklist2 in_queue, Worklist2 out_queue, int depth) {
	//expandByCta(m, row_offsets, column_indices, dist, in_queue, out_queue, depth);
	//expandByWarp(m, row_offsets, column_indices, dist, in_queue, out_queue, depth);
	unsigned id = blockIdx.x * blockDim.x + threadIdx.x;
	int vertex;
	const int SCRATCHSIZE = BLKSIZE;
	__shared__ BlockScan::TempStorage temp_storage;
	__shared__ int gather_offsets[SCRATCHSIZE];
	gather_offsets[threadIdx.x] = 0;
	int neighbor_size = 0;
	int neighbor_offset = 0;
	int scratch_offset = 0;
	int total_edges = 0;
	if(in_queue.pop_id(id, vertex)) {
		if(vertex != -1) {
			neighbor_offset = row_offsets[vertex];
			neighbor_size = row_offsets[vertex+1] - neighbor_offset;
		}
	}
	BlockScan(temp_storage).ExclusiveSum(neighbor_size, scratch_offset, total_edges);
	int done = 0;
	int neighbors_done = 0;
	while(total_edges > 0) {
		__syncthreads();
		int i;
		for(i = 0; neighbors_done + i < neighbor_size && (scratch_offset + i - done) < SCRATCHSIZE; i++) {
			gather_offsets[scratch_offset + i - done] = neighbor_offset + neighbors_done + i;
		}
		neighbors_done += i;
		scratch_offset += i;
		__syncthreads();
		//int ncnt = 0;
		int dst = 0;
		int edge = gather_offsets[threadIdx.x];
		if(threadIdx.x < total_edges) {
			dst = column_indices[edge];
			assert(dst < m);
			if(dist[dst] == MYINFINITY) {
				dist[dst] = depth;
				//ncnt = 1;
				out_queue.push(dst);
			}
		}
		//out_queue.push_1item<BlockScan>(ncnt, dst, BLKSIZE);
		total_edges -= BLKSIZE;
		done += BLKSIZE;
	}
}

__global__ void insert(int source, Worklist2 in_queue) {
	unsigned id = blockIdx.x * blockDim.x + threadIdx.x;
	if(id == 0) {
		in_queue.push(source);
	}
	return;
}

void BFSSolver(int m, int nnz, int source, int *in_row_offsets, int *in_column_indices, int *h_row_offsets, int *h_column_indices, int *h_degree, DistT *h_dist) {
	DistT zero = 0;
	int iter = 0;
	Timer t;
	int nthreads = BLKSIZE;
	int nblocks = (m - 1) / nthreads + 1;

	int *d_row_offsets, *d_column_indices;
	CUDA_SAFE_CALL(hipMalloc((void **)&d_row_offsets, (m + 1) * sizeof(int)));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_column_indices, nnz * sizeof(int)));
	CUDA_SAFE_CALL(hipMemcpy(d_row_offsets, h_row_offsets, (m + 1) * sizeof(int), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_column_indices, h_column_indices, nnz * sizeof(int), hipMemcpyHostToDevice));
	DistT * d_dist;
	CUDA_SAFE_CALL(hipMalloc((void **)&d_dist, m * sizeof(DistT)));
	CUDA_SAFE_CALL(hipMemcpy(d_dist, h_dist, m * sizeof(DistT), hipMemcpyHostToDevice));

	//initialize <<<nblocks, nthreads>>> (m, d_dist);
	//CudaTest("initializing failed");
	CUDA_SAFE_CALL(hipMemcpy(&d_dist[source], &zero, sizeof(zero), hipMemcpyHostToDevice));
	Worklist2 queue1(nnz), queue2(nnz);
	Worklist2 *in_frontier = &queue1, *out_frontier = &queue2;
	int nitems = 1;
	t.Start();
	insert<<<1, BLKSIZE>>>(source, *in_frontier);
	nitems = in_frontier->nitems();
	do {
		++ iter;
		nblocks = (nitems + BLKSIZE - 1) / BLKSIZE; 
		//printf("iteration=%d, nblocks=%d, nthreads=%d, wlsz=%d\n", iter, nblocks, BLKSIZE, nitems);
		bfs_kernel<<<nblocks, BLKSIZE>>>(m, d_row_offsets, d_column_indices, d_dist, *in_frontier, *out_frontier, iter);
		CudaTest("solving failed");
		nitems = out_frontier->nitems();
		Worklist2 *tmp = in_frontier;
		in_frontier = out_frontier;
		out_frontier = tmp;
		out_frontier->reset();
	} while(nitems > 0);
	CUDA_SAFE_CALL(hipDeviceSynchronize());
	t.Stop();
	printf("\titerations = %d.\n", iter);
	printf("\truntime [%s] = %f ms.\n", BFS_VARIANT, t.Millisecs());

	CUDA_SAFE_CALL(hipMemcpy(h_dist, d_dist, m * sizeof(DistT), hipMemcpyDeviceToHost));
	CUDA_SAFE_CALL(hipFree(d_row_offsets));
	CUDA_SAFE_CALL(hipFree(d_column_indices));
	CUDA_SAFE_CALL(hipFree(d_dist));
	return;
}
