// Copyright 2016, National University of Defense Technology
// Authors: Xuhao Chen <cxh@illinois.edu> and Pingfan Li <lipingfan@163.com>
#define BFS_VARIANT "merrill"
#include <hipcub/hipcub.hpp>
#include "bfs.h"
#include "gbar.h"
#include "cuda_launch_config.hpp"
#include "cutil_subset.h"
#include "timer.h"
#include <b40c_test_util.h>
#include <b40c/graph/builder/dimacs.cuh>
#include <b40c/graph/bfs/csr_problem.cuh>
#include <b40c/graph/csr_graph.cuh>
#include <b40c/graph/bfs/enactor_hybrid.cuh>
#include <b40c/graph/bfs/enactor_two_phase.cuh>
using namespace b40c;
using namespace graph;

void BFSSolver(int m, int nnz, int *in_row_offsets, int *in_column_indices, int *h_row_offsets, int *h_column_indices, int *h_degree, DistT *h_dist) {
	printf("BFS data-driven Merrill's version\n");
	typedef int VertexId;
	typedef unsigned Value;
	typedef int SizeT;
	int *d_row_offsets, *d_column_indices;
	DistT *d_dist;
	CUDA_SAFE_CALL(hipMalloc((void **)&d_row_offsets, (m + 1) * sizeof(int)));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_column_indices, nnz * sizeof(int)));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_dist, m * sizeof(DistT)));
	CUDA_SAFE_CALL(hipMemcpy(d_row_offsets, h_row_offsets, (m + 1) * sizeof(int), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_column_indices, h_column_indices, nnz * sizeof(int), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_dist, h_dist, m * sizeof(DistT), hipMemcpyHostToDevice));

	graph::CsrGraph<VertexId, Value, SizeT> csr_graph;
	csr_graph.FromScratch<true>(m, nnz);
	CUDA_SAFE_CALL(hipMemcpy(csr_graph.row_offsets, d_row_offsets, sizeof(SizeT) * (m + 1), hipMemcpyDeviceToHost));
	CUDA_SAFE_CALL(hipMemcpy(csr_graph.column_indices, d_column_indices, sizeof(VertexId) * nnz, hipMemcpyDeviceToHost));

	typedef bfs::CsrProblem<VertexId, SizeT, false> CsrProblem;
	//bfs::EnactorTwoPhase<false> two_phase(false);
	bfs::EnactorHybrid<false> hybrid(false);
	CsrProblem csr_problem;
	if (csr_problem.FromHostProblem(false, csr_graph.nodes, csr_graph.edges, csr_graph.column_indices, csr_graph.row_offsets, 1)) exit(1);
	hipError_t	retval = hipSuccess;
	Timer t;
	t.Start();
	if (retval = csr_problem.Reset(hybrid.GetFrontierType(), 1.3))
	//if (retval = csr_problem.Reset(two_phase.GetFrontierType(), 1.3))
		return;
	if (retval = hybrid.EnactSearch(csr_problem, 0)) {
	//if (retval = two_phase.EnactIterativeSearch(csr_problem, 0)) {
		if (retval && (retval != hipErrorInvalidDeviceFunction)) {
			exit(1);
		}
	}
	CUDA_SAFE_CALL(hipDeviceSynchronize());
	t.Stop();
	printf("\truntime [%s] = %f ms.\n", BFS_VARIANT, t.Millisecs());
	CUDA_SAFE_CALL(hipMemcpy(d_dist, h_dist, m * sizeof(DistT), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipFree(d_row_offsets));
	CUDA_SAFE_CALL(hipFree(d_column_indices));
	CUDA_SAFE_CALL(hipFree(d_dist));
}
