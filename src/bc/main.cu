// Copyright 2016, National University of Defense Technology
// Authors: Xuhao Chen <cxh@illinois.edu>
#include <stdio.h>
using namespace std;
#include "common.h"
#include "graph_io.h"
#include "variants.h"
#include "verifier.h"

int main(int argc, char *argv[]) {
	printf("Betweenness Centrality with CUDA by Xuhao Chen\n");
	if (argc < 2) {
		printf("Usage: %s <graph> [device(0/1)]\n", argv[0]);
		exit(1);
	}
	int m, nnz, *h_row_offsets = NULL, *h_column_indices = NULL, *h_degree = NULL;
	W_TYPE *h_weight = NULL;
	read_graph(argc, argv, m, nnz, h_row_offsets, h_column_indices, h_degree, h_weight, false);
	print_device_info(argc, argv);

	int *d_row_offsets, *d_column_indices, *d_degree;
	CUDA_SAFE_CALL(hipMalloc((void **)&d_row_offsets, (m + 1) * sizeof(int)));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_column_indices, nnz * sizeof(int)));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_degree, m * sizeof(int)));
	CUDA_SAFE_CALL(hipMemcpy(d_row_offsets, h_row_offsets, (m + 1) * sizeof(int), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_column_indices, h_column_indices, nnz * sizeof(int), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_degree, h_degree, m * sizeof(int), hipMemcpyHostToDevice));

	ScoreT *h_scores = (ScoreT *)malloc(m * sizeof(ScoreT));
	ScoreT *d_scores;
	CUDA_SAFE_CALL(hipMalloc((void **)&d_scores, sizeof(ScoreT) * m));
	BCSolver(m, nnz, d_row_offsets, d_column_indices, d_scores);
	CUDA_SAFE_CALL(hipMemcpy(h_scores, d_scores, sizeof(ScoreT) * m, hipMemcpyDeviceToHost));
	//for (int i = 0; i < 10; i++) printf("scores[%d] = %.8f\n", i, h_scores[i]);
	BCVerifier(m, h_row_offsets, h_column_indices, 1, h_scores);
	CUDA_SAFE_CALL(hipFree(d_row_offsets));
	CUDA_SAFE_CALL(hipFree(d_column_indices));
	CUDA_SAFE_CALL(hipFree(d_degree));
	free(h_row_offsets);
	free(h_column_indices);
	free(h_degree);
	return 0;
}
