// Copyright 2016, National University of Defense Technology
// Authors: Xuhao Chen <cxh@illinois.edu> and Pingfan Li <lipingfan@163.com>
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <fstream>
#include <sstream>
#include <string>
#include <vector>
#include <set>
using namespace std;
#include "common.h"
#include "graph_io.h"
#include "variants.h"

int main(int argc, char *argv[]) {
	printf("Breadth-first Search with CUDA by Xuhao Chen\n");
	if (argc < 2) {
		printf("Usage: %s <graph> [device(0/1)]\n", argv[0]);
		exit(1);
	}
	int m, nnz, *h_row_offsets = NULL, *h_column_indices = NULL;
	foru *h_weight = NULL;
	if (strstr(argv[1], ".mtx"))
		mtx2csr(argv[1], m, nnz, h_row_offsets, h_column_indices, h_weight);
	else if (strstr(argv[1], ".graph"))
		graph2csr(argv[1], m, nnz, h_row_offsets, h_column_indices, h_weight);
	else if (strstr(argv[1], ".gr"))
		gr2csr(argv[1], m, nnz, h_row_offsets, h_column_indices, h_weight);
	else { printf("Unrecognizable input file format\n"); exit(0); }

	int device = 0;
	if (argc > 2) device = atoi(argv[2]);
	assert(device == 0 || device == 1);
	int deviceCount = 0;
	CUDA_SAFE_CALL(hipGetDeviceCount(&deviceCount));
	CUDA_SAFE_CALL(hipSetDevice(device));
	hipDeviceProp_t deviceProp;
	CUDA_SAFE_CALL(hipGetDeviceProperties(&deviceProp, device));
	int nSM = deviceProp.multiProcessorCount;
	fprintf(stdout, "Found %d devices, using device %d (%s), compute capability %d.%d, cores %d*%d.\n", 
			deviceCount, device, deviceProp.name, deviceProp.major, deviceProp.minor, nSM, ConvertSMVer2Cores(deviceProp.major, deviceProp.minor));

	foru *h_dist = (foru *) malloc(m * sizeof(foru));
	for(int i = 0; i < m; i ++) {
		h_dist[i] = MYINFINITY;
	}
	foru * d_dist;
	CUDA_SAFE_CALL(hipMalloc((void **)&d_dist, m * sizeof(foru)));
	CUDA_SAFE_CALL(hipMemcpy(d_dist, h_dist, m * sizeof(foru), hipMemcpyHostToDevice));
	int *d_row_offsets, *d_column_indices;
	CUDA_SAFE_CALL(hipMalloc((void **)&d_row_offsets, (m + 1) * sizeof(int)));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_column_indices, nnz * sizeof(int)));
	CUDA_SAFE_CALL(hipMemcpy(d_row_offsets, h_row_offsets, (m + 1) * sizeof(int), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_column_indices, h_column_indices, nnz * sizeof(int), hipMemcpyHostToDevice));
	bfs(m, nnz, d_row_offsets, d_column_indices, d_dist, nSM);
	CUDA_SAFE_CALL(hipMemcpy(h_dist, d_dist, m * sizeof(foru), hipMemcpyDeviceToHost));
	printf("Verifying...\n");
	unsigned h_nerr = 0;
	verify(m, h_dist, h_row_offsets, h_column_indices, h_weight, &h_nerr);
	printf("\tNumber of errors = %d.\n", h_nerr);
	write_solution("bfs-out.txt", m, h_dist);
	CUDA_SAFE_CALL(hipFree(d_row_offsets));
	CUDA_SAFE_CALL(hipFree(d_column_indices));
	CUDA_SAFE_CALL(hipFree(d_dist));
	free(h_row_offsets);
	free(h_column_indices);
	free(h_dist);
	return 0;
}
